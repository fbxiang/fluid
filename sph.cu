#include "hip/hip_runtime.h"
#include "sph.cuh"
#include "sph_math.h"
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include "marching_cube_table.h"
#include "random.h"
#include "profiler.h"

#define N_THREADS 1024
#define CUDA_CHECK_RETURN(value)                                               \
  {                                                                            \
    hipError_t _m_cudaStat = value;                                           \
    if (_m_cudaStat != hipSuccess) {                                          \
      fprintf(stderr, "Error %s at line %d in file %s\n",                      \
              hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);            \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  }

namespace sph {

__constant__ SolverParams params;
SolverParams h_params;
__constant__ FluidDomain domain;
FluidDomain h_domain;
__constant__ glm::ivec3 grid_size;
glm::ivec3 h_grid_size;

__device__ int *grid;
int *d_grid;

__device__ int *grid_start_idx;
int *d_grid_start_idx;

__device__ glm::vec3 *positions;
glm::vec3 *d_positions;

__device__ glm::vec3 *velocities;
glm::vec3 *d_velocities;

__device__ glm::vec3 *force;
glm::vec3 *d_force;

__device__ int *cell_idx;
int *d_cell_idx;

__device__ int *sorted_particle_idx;
int *d_sorted_particle_idx;

__device__ float *rho;
float *d_rho;

__device__ float* tmp_float;
float* d_tmp_float;

__device__ glm::vec3 *velocities_pred;
glm::vec3 *d_velocities_pred;

__device__ glm::vec3 *positions_pred;
glm::vec3 *d_positions_pred;

__device__ float* pressure;
float* d_pressure;

__device__ glm::vec3 *non_pressure_force;  // used in PCISPH
glm::vec3 *d_non_pressure_force;

__device__ glm::vec3 *pressure_force;  // used in PCISPH
glm::vec3 *d_pressure_force;

__device__ float dt;

__device__ glm::vec3 *color_grad;  // particle normal
glm::vec3 *d_color_grad;

__device__ float *air_potential;
float *d_air_potential;

__device__ float *air_energy;
float *d_air_energy;

__device__ glm::vec3 *visual_debug;
glm::vec3 *d_visual_debug;



int max_num_particles;
int num_particles;
int num_cells;

void cuda_clear(void* d_field, size_t elem_size) {
  hipMemset(d_field, 0, num_particles * elem_size);
}

__device__ float psi(float I, float tau_min, float tau_max) {
  return (min(I, tau_max) - min(I, tau_min)) / (tau_max - tau_min);
}

#define FOR_NEIGHBORS(...)                                                     \
  {                                                                            \
    float c = params.cell_size;                                                \
    glm::ivec3 gpos = pos2grid_pos(positions[i]);                              \
    for (int x = -1; x <= 1; ++x) {                                            \
      if (x == -1 && gpos.x == 0 || x == 1 && gpos.x == grid_size.x - 1)       \
        continue;                                                              \
      for (int y = -1; y <= 1; ++y) {                                          \
        if (y == -1 && gpos.y == 0 || y == 1 && gpos.y == grid_size.y - 1)     \
          continue;                                                            \
        for (int z = -1; z <= 1; ++z) {                                        \
          if (z == -1 && gpos.z == 0 || z == 1 && gpos.z == grid_size.y - 1)   \
            continue;                                                          \
          int cell_idx = grid_pos2cell_idx(                                    \
              glm::ivec3(gpos.x + x, gpos.y + y, gpos.z + z));                 \
          for (int k = grid_start_idx[cell_idx], e = k + grid[cell_idx];       \
               k < e; ++k) {                                                   \
            int j = sorted_particle_idx[k];                                    \
            glm::vec3 r = positions[i] - positions[j];                         \
            if (glm::dot(r, r) < c * c) {                                      \
              __VA_ARGS__                                                      \
            }                                                                  \
          }                                                                    \
        }                                                                      \
      }                                                                        \
    }                                                                          \
  }


__device__ glm::ivec3 pos2grid_pos(glm::vec3 pos) {
  return (pos - domain.corner) / params.cell_size;
}

__device__ int grid_pos2cell_idx(glm::ivec3 gpos) {
  return gpos.x * grid_size.y * grid_size.z + gpos.y * grid_size.z + gpos.z;
}

__device__ int pos2cell_idx(glm::vec3 pos) {
  return grid_pos2cell_idx(pos2grid_pos(pos));
}

__device__ glm::vec3 gradient(float *quantity, int i) {
  glm::vec3 value = {0,0,0};

  FOR_NEIGHBORS(
      if (i == j) { continue; }
      value += params.particle_mass *
      (quantity[i] / rho[i] / rho[i] + quantity[j] / rho[j] / rho[j]) *
      dw_ij(positions[i], positions[j], params.particle_size););
  return rho[i] * value;
}


__device__ glm::vec3 laplacian(glm::vec3 *quantity, int i) {
  float h = params.particle_size;
  glm::vec3 value = {0,0,0};
  FOR_NEIGHBORS(
      if (i == j) {continue;}
      glm::vec3 xij = positions[i] - positions[j];
      value += (quantity[i] - quantity[j]) *
      (params.particle_mass / rho[j] * glm::dot(xij, dw_ij(positions[i], positions[j], h)) /
       (glm::dot(xij, xij) + 0.01f * h * h));
                );
  return 2.f * value;
}

__device__ float divergence(glm::vec3 *vec, int i) {
  float value = 0;
  FOR_NEIGHBORS(if (i == j) { continue; } value +=
                params.particle_mass *
                glm::dot(vec[i] - vec[j], dw_ij(positions[i], positions[j],
                                                params.particle_size)););
  value /= -rho[i];
  return value;
}


__global__ void _update_cell_idx(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    cell_idx[i] = pos2cell_idx(positions[i]);
    atomicAdd(&grid[cell_idx[i]], 1);
  }
}
__global__ void _update_sorted_particle_idx(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    int idx = atomicAdd(&grid_start_idx[cell_idx[i]], 1);
    sorted_particle_idx[idx] = i;
  }
}
__global__ void _restore_start_idx(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    atomicSub(&grid_start_idx[cell_idx[i]], 1);
  }
}

__global__ void _velocity_to_speed(float* d_speed, glm::vec3 *d_vel, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    d_speed[i] = glm::length(d_vel[i]);
  }
}

bool add_particles(glm::vec3 *in_positions, int size) {
  if (num_particles + size > max_num_particles)
    return false;
  hipMemcpy(&d_positions[num_particles], in_positions, size * sizeof(glm::vec3),
             hipMemcpyHostToDevice);
  hipMemset(&d_velocities[num_particles], 0, size * sizeof(glm::vec3));
  num_particles += size;
  return true;
}

bool get_particles(glm::vec3 *out_positions, int size) {
  if (size <= 0 || size > num_particles) {
    return false;
  }
  hipMemcpy(out_positions, d_positions, size * sizeof(glm::vec3),
             hipMemcpyDeviceToHost);
  return true;
}

// all solvers
void update_neighbors() {
  // clear grid
  CUDA_CHECK_RETURN(hipMemset(d_grid, 0, num_cells * sizeof(int)));

  // update cell idx
  _update_cell_idx<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(
      num_particles);
  // update sorted particle idx
  // 1. scan
  thrust::device_ptr<int> grid_ptr = thrust::device_pointer_cast(d_grid);
  thrust::device_ptr<int> grid_start_idx_ptr =
      thrust::device_pointer_cast(d_grid_start_idx);
  thrust::exclusive_scan(grid_ptr, grid_ptr + num_cells, grid_start_idx_ptr);
  // 2. update
  _update_sorted_particle_idx<<<(num_particles + N_THREADS - 1) / N_THREADS,
                                N_THREADS>>>(num_particles);
  // 3. restore
  _restore_start_idx<<<(num_particles + N_THREADS - 1) / N_THREADS,
                       N_THREADS>>>(num_particles);
}

// update dt using velocities
float update_dt_by_CFL() {
  _velocity_to_speed<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(d_tmp_float, d_velocities, num_particles);
  thrust::device_ptr<float> speed_ptr = thrust::device_pointer_cast(d_tmp_float);
  float max_speed = *thrust::max_element(speed_ptr, speed_ptr + num_particles);
  float dt = 0.2 * h_params.particle_size / max_speed;
  if (dt > 0.003) dt = 0.003;
  if (dt < 0.00005) dt = 0.00005;
  hipMemcpyToSymbol(HIP_SYMBOL(sph::dt), &dt, sizeof(float));
  return dt;
}

// update dt using velocities_pred
float update_dt_by_CFL_pred() {
  _velocity_to_speed<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(d_tmp_float, d_velocities_pred, num_particles);
  thrust::device_ptr<float> speed_ptr = thrust::device_pointer_cast(d_tmp_float);
  float max_speed = *thrust::max_element(speed_ptr, speed_ptr + num_particles);
  float dt = 0.4 * h_params.particle_size / max_speed;
  if (dt > 0.01) dt = 0.01;
  if (dt < 0.00005) dt = 0.00005;
  hipMemcpyToSymbol(HIP_SYMBOL(sph::dt), &dt, sizeof(float));
  return dt;
}


__global__ void _update_density(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float value = 0;
    FOR_NEIGHBORS(
        value += params.particle_mass * w_ij(positions[i], positions[j], params.particle_size);
                 );
    rho[i] = value;
  }
}
// update rho using positions
void update_density() {
  _update_density<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}


__global__ void _update_density_increment_pressure(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float rho_pred = 0;
    FOR_NEIGHBORS(
        rho_pred += params.particle_mass * w_ij(positions_pred[i], positions_pred[j], params.particle_size);
                 );

    // update pressure
    float rho_err = rho_pred - params.rest_density;
    float beta = dt*dt*params.particle_mass*params.particle_mass*2.f/(params.rest_density * params.rest_density);
    glm::vec3 vacc = {0, 0, 0};
    float sacc = 0;
    FOR_NEIGHBORS(
        if (i == j) continue;
        glm::vec3 dw = dw_ij(positions_pred[i], positions_pred[j], params.particle_size);
        vacc += dw;
        sacc += glm::dot(dw, dw);
                  );
    pressure[i] += rho_err / (beta * (glm::dot(vacc, vacc) + sacc));
    if (pressure[i] < 0) pressure[i] = 0;
  }
}
// increment pressure using rho_pred computed from positions_pred
void update_density_increment_pressure() {
  _update_density_increment_pressure<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

__global__ void _update_pressure_force(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    pressure_force[i] = -params.particle_mass / rho[i] * gradient(pressure, i);
    force[i] = pressure_force[i] + non_pressure_force[i]; 
  }
}
// pci solver, update pressure force and net-force from current pressure
void update_pressure_force() {
  _update_pressure_force<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

__global__ void _update_pressure(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    pressure[i] = params.k * params.rest_density / params.gamma *
                  (powf(rho[i] / params.rest_density, params.gamma) - 1.f);
  }
}
__global__ void _update_all_forces(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    glm::vec3 f_pressure = -params.particle_mass / rho[i] * gradient(pressure, i);
    glm::vec3 f_viscosity =
        params.particle_mass * params.viscosity * laplacian(velocities, i);
    glm::vec3 f_gravity = params.particle_mass * params.g;
    force[i] = f_pressure + f_viscosity + f_gravity;
  }
}
// only in regular solver, compute force directly from rho
void update_all_forces() {
  _update_pressure<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
  _update_all_forces<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}


// regular solver
__global__ void _update_velocity_position(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    velocities[i] += sph::dt * force[i] / params.particle_mass;
    positions[i] += sph::dt * velocities[i];

    if (positions[i].x < domain.corner.x + params.eps) {
      velocities[i].x = 0.f;
      positions[i].x = domain.corner.x + params.eps;
    }
    if (positions[i].y < domain.corner.y + params.eps) {
      velocities[i].y = 0.f;
      positions[i].y = domain.corner.y + params.eps;
    }
    if (positions[i].z < domain.corner.z + params.eps) {
      velocities[i].z = 0.f;
      positions[i].z = domain.corner.z + params.eps;
    }

    if (positions[i].x >= domain.corner.x + domain.size.x - params.eps) {
      velocities[i].x = 0.f;
      positions[i].x = domain.corner.x + domain.size.x - params.eps;
    }
    if (positions[i].y >= domain.corner.y + domain.size.y - params.eps) {
      velocities[i].y = 0.f;
      positions[i].y = domain.corner.y + domain.size.y - params.eps;
    }
    if (positions[i].z >= domain.corner.z + domain.size.z - params.eps) {
      velocities[i].z = 0.f;
      positions[i].z = domain.corner.z + domain.size.z - params.eps;
    }
  }
}

__global__ void _update_color_grad(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    glm::vec3 value;
    FOR_NEIGHBORS(
        if (i == j) continue;
        value += params.particle_mass / rho[j] * dw_ij(positions[i], positions[j], params.particle_size);
                  );
    color_grad[i] = value;
  }
}

__global__ void _update_visual(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    float v_diff = 0;
    float curvature = 0;
    glm::vec3 ni = glm::normalize(color_grad[i]);
    FOR_NEIGHBORS(
        if (i == j) continue;
        // v_diff
        glm::vec3 vij = velocities[i]-velocities[j];
        glm::vec3 xij = positions[i]-positions[j];
        float kn = kernel2(glm::length(xij) / params.particle_size);
        if (glm::length(vij) > 1e-6 && glm::length(xij) > 1e-6) {
          v_diff += glm::length(vij) * (1 - glm::dot(glm::normalize(vij), glm::normalize(xij))) * kn;
        }

        glm::vec3 nj = glm::normalize(color_grad[j]);
        curvature += (1 - glm::dot(ni, nj)) * kn * (glm::dot(xij, ni) > 0);
                  );
    glm::vec3 normal = glm::normalize(color_grad[i]);
    glm::vec3 vel_dir = glm::normalize(velocities[i]);
    bool crest = glm::length(color_grad[i]) > 10 && glm::dot(normal, vel_dir) > 0.6;

    float energy = glm::dot(velocities[i], velocities[i]);

    // v_diff
    float Ita = psi(v_diff, 10, 40);
    // curvature * crest
    float Iwc = psi(curvature * crest, 2, 8);
    // energy
    float Ik = psi(energy, 0, 1);
    float potential = Ik * (3000 * Ita + 1000 * Iwc) * dt;
    air_potential[i] = potential;
    air_energy[i] = Ik;

    float test = potential / dt;
    visual_debug[i] = glm::vec3(1, 1-test, 1-test);
  }
}

void update_velocity_position() {
  _update_velocity_position<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
  _update_color_grad<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
  _update_visual<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

__global__ void _update_non_pressure_forces(int n) {
  float E = 0.01;
  float F = 0.1;
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    glm::vec3 f_boundary = {0,0,0};

    if (positions[i].x < domain.corner.x + E) {
      f_boundary.x += F;
    }
    if (positions[i].y < domain.corner.y + E) {
      f_boundary.y += F;
    }
    if (positions[i].z < domain.corner.z + E) {
      f_boundary.z += F;
    }
    if (positions[i].x >= domain.corner.x + domain.size.x - E) {
      f_boundary.x -= F;
    }
    if (positions[i].y >= domain.corner.y + domain.size.y - E) {
      f_boundary.y -= F;
    }
    if (positions[i].z >= domain.corner.z + domain.size.z - E) {
      f_boundary.z -= F;
    }

    glm::vec3 f_gravity = params.particle_mass * params.g;

    glm::vec3 f_viscosity =
        params.particle_mass * params.viscosity * laplacian(velocities, i);

    non_pressure_force[i] = f_gravity + f_boundary + f_viscosity;
    force[i] = non_pressure_force[i] + pressure_force[i];
  }
}
void update_non_pressure_forces() {
  _update_non_pressure_forces<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

__global__ void _update_positions(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    positions[i] += sph::dt * velocities_pred[i];

    if (positions[i].x < domain.corner.x + params.eps) {
      velocities[i].x = 0.f;
      positions[i].x = domain.corner.x + params.eps;
    }
    if (positions[i].y < domain.corner.y + params.eps) {
      velocities[i].y = 0.f;
      positions[i].y = domain.corner.y + params.eps;
    }
    if (positions[i].z < domain.corner.z + params.eps) {
      velocities[i].z = 0.f;
      positions[i].z = domain.corner.z + params.eps;
    }

    if (positions[i].x >= domain.corner.x + domain.size.x - params.eps) {
      velocities[i].x = 0.f;
      positions[i].x = domain.corner.x + domain.size.x - params.eps;
    }
    if (positions[i].y >= domain.corner.y + domain.size.y - params.eps) {
      velocities[i].y = 0.f;
      positions[i].y = domain.corner.y + domain.size.y - params.eps;
    }
    if (positions[i].z >= domain.corner.z + domain.size.z - params.eps) {
      velocities[i].z = 0.f;
      positions[i].z = domain.corner.z + domain.size.z - params.eps;
    }
  }
}
void update_positions() {
  _update_positions<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

__global__ void _update_velocity_pred(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    velocities_pred[i] = velocities[i] + dt * non_pressure_force[i] / params.particle_mass;
  }
}
/* Predict velocity only */
void update_velocity_pred() {
  _update_velocity_pred<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

__global__ void _update_velocity_position_pred(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    velocities_pred[i] = velocities[i] + sph::dt * force[i] / params.particle_mass;
    positions_pred[i] = positions[i] + sph::dt * velocities_pred[i];

    if (positions_pred[i].x < domain.corner.x + params.eps) {
      velocities_pred[i].x = 0.f;
      positions_pred[i].x = domain.corner.x + params.eps;
    }
    if (positions_pred[i].y < domain.corner.y + params.eps) {
      velocities_pred[i].y = 0.f;
      positions_pred[i].y = domain.corner.y + params.eps;
    }
    if (positions_pred[i].z < domain.corner.z + params.eps) {
      velocities_pred[i].z = 0.f;
      positions_pred[i].z = domain.corner.z + params.eps;
    }

    if (positions_pred[i].x >= domain.corner.x + domain.size.x - params.eps) {
      velocities_pred[i].x = 0.f;
      positions_pred[i].x = domain.corner.x + domain.size.x - params.eps;
    }
    if (positions_pred[i].y >= domain.corner.y + domain.size.y - params.eps) {
      velocities_pred[i].y = 0.f;
      positions_pred[i].y = domain.corner.y + domain.size.y - params.eps;
    }
    if (positions_pred[i].z >= domain.corner.z + domain.size.z - params.eps) {
      velocities_pred[i].z = 0.f;
      positions_pred[i].z = domain.corner.z + domain.size.z - params.eps;
    }
  }
}
/* Predict both velocity and position */
void update_velocity_position_pred() {
  _update_velocity_position_pred<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);
}

void update_velocity() {
  hipMemcpy(d_velocities, d_velocities_pred,
             num_particles * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);
}

float step_regular() {
  profiler::start("neighbors");
  update_neighbors();
  profiler::stop("neighbors");

  profiler::start("density_pressure");
  float dt = update_dt_by_CFL();
  update_density();
  update_all_forces();
  profiler::stop("density_pressure");

  profiler::start("integration");
  update_velocity_position();
  profiler::stop("integration");

  return dt;
}

float pci_step() {
  update_neighbors();

  cuda_clear(d_pressure_force, sizeof(glm::vec3));
  cuda_clear(d_pressure, sizeof(float));
  update_density();
  update_non_pressure_forces();

  update_velocity_position_pred();
  float dt = update_dt_by_CFL_pred();

  for (int iter = 0; iter < 5; ++iter) {
    update_density_increment_pressure();
    update_pressure_force();
    update_velocity_position_pred();
  }
  update_velocity_position();

  return dt;
}

__global__ void _update_debug_points(float* vbo, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    vbo[6 * i    ] = positions[i].x;
    vbo[6 * i + 1] = positions[i].y;
    vbo[6 * i + 2] = positions[i].z;

    vbo[6 * i + 3] = visual_debug[i].x;
    vbo[6 * i + 4] = visual_debug[i].y;
    vbo[6 * i + 5] = visual_debug[i].z;
  }
}

void update_debug_points(float* vbo) {
  _update_debug_points<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(vbo, num_particles);
}


void init(const SolverParams &in_params, const FluidDomain &in_domain,
          const int max_num_particles) {
  hipMemcpyToSymbol(HIP_SYMBOL(sph::params), &in_params, sizeof(SolverParams));
  h_params = in_params;
  hipMemcpyToSymbol(HIP_SYMBOL(sph::domain), &in_domain, sizeof(FluidDomain));
  h_domain = in_domain;

  glm::ivec3 grid_size = glm::ivec3(ceil(in_domain.size.x / in_params.cell_size),
                                    ceil(in_domain.size.y / in_params.cell_size),
                                    ceil(in_domain.size.z / in_params.cell_size));
  h_grid_size = grid_size;
  hipMemcpyToSymbol(HIP_SYMBOL(sph::grid_size), &grid_size, sizeof(glm::ivec3));

  sph::num_cells = grid_size.x * grid_size.y * grid_size.z;
  sph::max_num_particles = max_num_particles;
  sph::num_particles = 0;

  // common quantities
  CUDA_CHECK_RETURN(hipMalloc(&d_grid, num_cells * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::grid), &d_grid, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_grid_start_idx, num_cells * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::grid_start_idx), &d_grid_start_idx,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_positions, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::positions), &d_positions, sizeof(void *),
                                       0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_velocities, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(
      sph::velocities), &d_velocities, sizeof(void *), 0, hipMemcpyHostToDevice));


  CUDA_CHECK_RETURN(hipMalloc(&d_force, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(
      sph::force), &d_force, sizeof(void *), 0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_pressure_force, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(
      sph::pressure_force), &d_pressure_force, sizeof(void *), 0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_non_pressure_force, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(
      sph::non_pressure_force), &d_non_pressure_force, sizeof(void *), 0, hipMemcpyHostToDevice));


  CUDA_CHECK_RETURN(hipMalloc(&d_cell_idx, max_num_particles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::cell_idx), &d_cell_idx, sizeof(void *),
                                       0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_sorted_particle_idx, max_num_particles * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::sorted_particle_idx), &d_sorted_particle_idx,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_rho, max_num_particles * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::rho), &d_rho, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_tmp_float, max_num_particles * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::tmp_float), &d_tmp_float, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));
  
  // regular solver
  CUDA_CHECK_RETURN(hipMalloc(&d_pressure, max_num_particles * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::pressure), &d_pressure, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_velocities_pred, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::velocities_pred), &d_velocities_pred, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_positions_pred, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::positions_pred), &d_positions_pred, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_color_grad, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::color_grad), &d_color_grad, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_air_potential, max_num_particles * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::air_potential), &d_air_potential, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_air_energy, max_num_particles * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::air_energy), &d_air_energy, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));


  CUDA_CHECK_RETURN(hipMalloc(&d_visual_debug, max_num_particles * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(sph::visual_debug), &d_visual_debug, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

}

int get_num_particles() {
  return num_particles;
}

void log() {
  printf("\n");
}
void log(const char* key) {
  printf("%s\n", key);
}
void log(const char* key, int value) {
  printf("%-30s %d\n", key, value);
}
void log(const char* key, float value) {
  printf("%-30s %.3f\n", key, value);
}
void log(const char* key, glm::vec3 value) {
  printf("%-30s %f %f %f\n", key, value.x, value.y, value.z);
}
void log(const char* key, glm::ivec3 value) {
  printf("%-30s %d %d %d\n", key, value.x, value.y, value.z);
}
void log_array(const char* key, int* array, int size) {
  printf("%s\n", key);
  int i = 0;
  while (i < size) {
    for (int j = 0; j < 10 && i < size; ++j) {
      printf("%6d ", array[i++]);
    }
  }
  printf("\n");
}

void print_summary() {
  log("Simulation State");
  log("Max size", max_num_particles);
  log("Current size", num_particles);
  log();

  log("Parameters");
  SolverParams h_params;
  hipMemcpyFromSymbol(&h_params, HIP_SYMBOL(sph::params), sizeof(SolverParams));
  log("Particle size", h_params.particle_size);
  log("Cell size", h_params.cell_size);
  log("Rest density", h_params.rest_density);
  log();

  log("Domain");
  FluidDomain h_domain;
  hipMemcpyFromSymbol(&h_domain, HIP_SYMBOL(sph::domain), sizeof(FluidDomain));
  log("Corner", h_domain.corner);
  log("Size", h_domain.size);

  glm::ivec3 h_grid_size;
  hipMemcpyFromSymbol(&h_grid_size, HIP_SYMBOL(sph::grid_size), sizeof(glm::ivec3));
  log("Grid size", h_grid_size);
  log("Number of cells", num_cells);
  log();
}


namespace visual {

#define VISUAL_FOR_NEIGHBORS(...)                                              \
  {                                                                            \
    float c = params.cell_size;                                                \
    glm::ivec3 gpos = pos2grid_pos(visual::positions[i]);                      \
    for (int x = -1; x <= 1; ++x) {                                            \
      if (x == -1 && gpos.x == 0 || x == 1 && gpos.x == grid_size.x - 1)       \
        continue;                                                              \
      for (int y = -1; y <= 1; ++y) {                                          \
        if (y == -1 && gpos.y == 0 || y == 1 && gpos.y == grid_size.y - 1)     \
          continue;                                                            \
        for (int z = -1; z <= 1; ++z) {                                        \
          if (z == -1 && gpos.z == 0 || z == 1 && gpos.z == grid_size.y - 1)   \
            continue;                                                          \
          int cell_idx = grid_pos2cell_idx(                                    \
              glm::ivec3(gpos.x + x, gpos.y + y, gpos.z + z));                 \
          for (int k = grid_start_idx[cell_idx], e = k + grid[cell_idx];       \
               k < e; ++k) {                                                   \
            int j = sorted_particle_idx[k];                                    \
            glm::vec3 r = visual::positions[i] - sph::positions[j];            \
            if (glm::dot(r, r) < c * c) {                                      \
              __VA_ARGS__                                                      \
            }                                                                  \
          }                                                                    \
        }                                                                      \
      }                                                                        \
    }                                                                          \
  }


__device__ glm::vec3 *positions;
glm::vec3 *d_positions;

__device__ glm::vec3 *velocities;
glm::vec3 *d_velocities;

__device__ float *life;
float *d_life;

__device__ int *visual_types;
int *d_visual_types;

__device__ unsigned int *rand_seeds;
unsigned int *d_rand_seeds;

__constant__ int max_visual = 100000;
constexpr int h_max_visual = 100000;
__device__ int visual_count = 0;
int h_visual_count = 0;

int get_num_visual_particles() {
  return h_visual_count;
}

// Call add_particle on each fluid particle to add diffuse particle
__device__ void add_visual_particle(glm::vec3 position, glm::vec3 velocity, float life) {
  int idx = atomicAdd(&visual_count, 1);
  if (visual_count > max_visual) {
    atomicAdd(&visual_count, -1);
    return;
  }
  visual::positions[idx] = position;
  visual::velocities[idx] = velocity;
  visual::life[idx] = life * 5 + rnd(rand_seeds[idx]) * 2;
}
__global__ void _add_visual_particles(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    if (rnd(rand_seeds[i]) < sph::air_potential[i]) {
      glm::vec3 p = sph::positions[i];
      glm::vec3 v = sph::velocities[i];

      glm::vec3 n = glm::normalize(v);
      glm::vec3 e1, e2;
      if (n.y < 0.9) {
        e1 = glm::normalize(glm::cross(glm::vec3(0,1,0), n));
        e2 = glm::cross(n, e1);
      } else {
        e1 = glm::normalize(glm::cross(glm::vec3(1,0,0), n));
        e2 = glm::cross(n, e1);
      }

      float r = rnd(rand_seeds[i]) * params.particle_size / 2.f;
      float h = rnd(rand_seeds[i]) * dt * glm::length(v);
      float theta = rnd(rand_seeds[i]) * M_PIf * 2.f;

      float dh = h;
      float dx = r * cos(theta);
      float dy = r * sin(theta);

      add_visual_particle(p + dx * e1 + dy * e2 + dh * n, v + dx * e1 + dy * e2, air_energy[i]);
    }
  }
}

void add_visual_particles() {
  // add visual particles
  _add_visual_particles<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles);

  // update host visual count
  CUDA_CHECK_RETURN(hipMemcpyFromSymbol(&h_visual_count, HIP_SYMBOL(visual_count), sizeof(int)));
}

__global__ void _update_visual_particles(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    glm::vec3 vf(0.f);
    float weights = 0;
    int count = 0;
    VISUAL_FOR_NEIGHBORS(
        ++count;
        float kn = kernel(glm::length(visual::positions[i] - sph::positions[j])/params.particle_size);
        weights += kn;
        vf += sph::velocities[j] * kn;
                         );
    vf /= weights;
    float kb = 0.2;
    float kd = 0.8;

    if (count < 12) {
      // spray
      visual_types[i] = 0;
      velocities[i] += params.g * dt;
    } else if (count > 35) {
      // bubble
      visual_types[i] = 1;
      velocities[i] += (-kb * params.g + kd * (vf-velocities[i])/dt) * dt;
    } else {
      // foam
      visual_types[i] = 2;
      velocities[i] = vf;
      life[i] -= dt;
    }
    positions[i] += velocities[i] * dt;

    // force boundary
    if (positions[i].x < domain.corner.x + params.eps) {
      velocities[i].x = 0.f;
      positions[i].x = domain.corner.x + params.eps;
    }
    if (positions[i].y < domain.corner.y + params.eps) {
      velocities[i].y = 0.f;
      positions[i].y = domain.corner.y + params.eps;
    }
    if (positions[i].z < domain.corner.z + params.eps) {
      velocities[i].z = 0.f;
      positions[i].z = domain.corner.z + params.eps;
    }

    if (positions[i].x >= domain.corner.x + domain.size.x - params.eps) {
      velocities[i].x = 0.f;
      positions[i].x = domain.corner.x + domain.size.x - params.eps;
    }
    if (positions[i].y >= domain.corner.y + domain.size.y - params.eps) {
      velocities[i].y = 0.f;
      positions[i].y = domain.corner.y + domain.size.y - params.eps;
    }
    if (positions[i].z >= domain.corner.z + domain.size.z - params.eps) {
      velocities[i].z = 0.f;
      positions[i].z = domain.corner.z + domain.size.z - params.eps;
    }
  }
}
void update_visual_particles() {
  if (h_visual_count <= 0) return;
  _update_visual_particles<<<(h_visual_count + N_THREADS - 1) / N_THREADS, N_THREADS>>>(h_visual_count);
}

__global__ void _init_rand_seeds(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    rand_seeds[i] = tea<16>(i, 0);
  }
}

struct _negative_pred : public thrust::unary_function<float,bool>
{
  __host__ __device__ float operator()(float x) { return x <= 0; }
};

void clear_visual_particles() {
  thrust::device_ptr<glm::vec3> positions_ptr = thrust::device_pointer_cast(d_positions);
  thrust::device_ptr<glm::vec3> velocities_ptr = thrust::device_pointer_cast(d_velocities);
  thrust::device_ptr<float> life_ptr = thrust::device_pointer_cast(d_life);
  if (h_visual_count > 0) {
    int new_count = thrust::remove_if(positions_ptr, positions_ptr + h_visual_count,
                                      life_ptr, _negative_pred()) - positions_ptr;
    thrust::remove_if(velocities_ptr, velocities_ptr + h_visual_count, life_ptr, _negative_pred());
    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(visual::visual_count), &new_count, sizeof(int)));
    h_visual_count = new_count;
  }
}

void visual_step() {
  add_visual_particles();
  update_visual_particles();
  clear_visual_particles();
}

void init() {
  CUDA_CHECK_RETURN(hipMalloc(&visual::d_positions, h_max_visual * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(visual::positions), &visual::d_positions,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&visual::d_velocities, h_max_visual * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(visual::velocities), &visual::d_velocities,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&visual::d_life, h_max_visual * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(visual::life), &visual::d_life,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&visual::d_visual_types, h_max_visual * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(visual::visual_types), &visual::d_visual_types,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&visual::d_rand_seeds, h_max_visual * sizeof(unsigned int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(visual::rand_seeds), &visual::d_rand_seeds,
                                       sizeof(void *), 0,
                                       hipMemcpyHostToDevice));
  _init_rand_seeds<<<(h_max_visual + N_THREADS - 1) / N_THREADS, N_THREADS>>>(h_max_visual);
}

__global__ void _update_debug_points(float* vbo, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    vbo[6 * i    ] = positions[i].x;
    vbo[6 * i + 1] = positions[i].y;
    vbo[6 * i + 2] = positions[i].z;

    if (visual_types[i] == 0) {
      vbo[6 * i + 3] = 1.f;
      vbo[6 * i + 4] = 1.f; 
      vbo[6 * i + 5] = 0.f;
    } else if (visual_types[i] == 1) {
      vbo[6 * i + 3] = 0.f;
      vbo[6 * i + 4] = 0.f; 
      vbo[6 * i + 5] = 1.f;
    } else if (visual_types[i] == 2) {
      vbo[6 * i + 3] = 1.f;
      vbo[6 * i + 4] = 0.f; 
      vbo[6 * i + 5] = 1.f;
    }
  }
}
void update_debug_points(float* vbo) {
  if (h_visual_count <= 0) return;
  _update_debug_points<<<(h_visual_count + N_THREADS - 1) / N_THREADS, N_THREADS>>>(vbo, h_visual_count);
}


__global__ void _update_visual_faces(float *vbo, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    float r = params.particle_size / 10;
    glm::vec3 ns[4] = {
      glm::normalize(glm::vec3(-1,1,-1)),
      glm::normalize(glm::vec3(-1,-1,1)),
      glm::normalize(glm::vec3(1,-1,-1)),
      glm::normalize(glm::vec3(1,1,1))
    };
    glm::vec3 vs[4] = {
      positions[i] + ns[1] * r,
      positions[i] + ns[2] * r,
      positions[i] + ns[3] * r,
      positions[i] + ns[4] * r
    };
    int arr[] = {0,2,1, 0,1,3, 0,3,2, 1,2,3};
    int j = 72 * i;
    for (int idx = 0; idx < 12; ++idx) {
      vbo[j++] = vs[arr[idx]].x;
      vbo[j++] = vs[arr[idx]].y;
      vbo[j++] = vs[arr[idx]].z;

      vbo[j++] = ns[arr[idx]].x;
      vbo[j++] = ns[arr[idx]].y;
      vbo[j++] = ns[arr[idx]].z;
    }
  }
}

void update_visual_faces(float* vbo) {
  if (h_visual_count <= 0) return;
  _update_visual_faces<<<(h_visual_count + N_THREADS - 1) / N_THREADS, N_THREADS>>>(vbo, h_visual_count);
}

}

namespace mc {
__constant__ glm::vec3 mc_corner;
glm::vec3 h_mc_corner;

__constant__ glm::vec3 mc_size;
glm::vec3 h_mc_size;

__constant__ glm::ivec3 grid_size;
glm::ivec3 h_grid_size;
__constant__ glm::ivec3 corner_size;
glm::ivec3 h_corner_size;
__constant__ float cell_size;
float h_cell_size;

__device__ int *grid_occupied;
int* d_grid_occupied;
__device__ float *corner_value;
float* d_corner_value;

__device__ glm::vec3 *faces;
glm::vec3 *d_faces;
__device__ glm::vec3 *face_normals;
glm::vec3 *d_face_normals;
__device__ int *num_faces;
int *d_num_faces;
__device__ int *grid_face_idx;  // used for stream compaction
int *d_grid_face_idx;


__device__ int get_corner_idx(glm::ivec3 v) {
  // TODO: get rid of this check?
  if (v.x < 0 || v.x >= corner_size.x ||
      v.y < 0 || v.y >= corner_size.y || 
      v.z < 0 || v.z >= corner_size.z) {
    return -1;
  }
  return v.x * corner_size.y * corner_size.z + v.y * corner_size.z + v.z;
}
__device__ int get_cell_idx(glm::ivec3 v) {
  return v.x * grid_size.y * grid_size.z + v.y * grid_size.z + v.z;
}
__device__ float get_corner_value(glm::ivec3 v) {
  int i = get_corner_idx(v);
  return i < 0 ? 0.f : corner_value[i];
}

__global__ void _update_grid_corners(int n, float radius) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    float r2 = radius * radius;
    glm::vec3 relative_position = (positions[i] - mc_corner);
    glm::vec3 start = relative_position - glm::vec3(radius);
    glm::vec3 end = relative_position + glm::vec3(radius);

    glm::ivec3 grid_start = glm::clamp(glm::ivec3(glm::ceil(start / cell_size)), glm::ivec3(0), corner_size);
    glm::ivec3 grid_end = glm::clamp(glm::ivec3(glm::ceil(end / cell_size)), glm::ivec3(0), corner_size);

    // TODO: look up table may be better
    for (int x = grid_start.x; x < grid_end.x; ++x) {
      for (int y = grid_start.y; y < grid_end.y; ++y) {
        for (int z = grid_start.z; z < grid_end.z; ++z) {
          glm::vec3 d = glm::vec3(x, y, z) * cell_size - relative_position;
          float d2 = glm::dot(d, d);
          if (d2 < r2) {
            int corner_idx = get_corner_idx(glm::ivec3(x,y,z));

            atomicAdd(&corner_value[corner_idx], kernel(sqrtf(d2) / params.particle_size));

            // TODO: the following may not be needed
            for (int x2 = -1; x2 <= 0; ++x2 ) {
              for (int y2 = -1; y2 <= 0; ++y2) {
                for (int z2 = -1; z2 <= 0; ++z2) {
                  glm::ivec3 cell_pos = glm::ivec3(x+x2, y+y2, z+z2);
                  if (cell_pos.x > 0 && cell_pos.x < grid_size.x &&
                      cell_pos.y > 0 && cell_pos.y < grid_size.y &&
                      cell_pos.z > 0 && cell_pos.z < grid_size.z) {
                    int cell_idx = get_cell_idx(cell_pos);
                    grid_occupied[cell_idx] = 1;
                  }
                }
              }
            }
          }
        }
      }
    }
  }
}

__device__ glm::vec3 get_vert_normal(glm::ivec3 v) {
  return -glm::normalize(glm::vec3(
      get_corner_value(v + glm::ivec3(1, 0, 0)) - get_corner_value(v + glm::ivec3(-1, 0, 0)),
      get_corner_value(v + glm::ivec3(0, 1, 0)) - get_corner_value(v + glm::ivec3(0, -1, 0)),
      get_corner_value(v + glm::ivec3(0, 0, 1)) - get_corner_value(v + glm::ivec3(0, 0, -1))));
}

__device__ glm::vec3 vertex_interp(float isolevel, glm::vec3 p0, glm::vec3 p1, float v0, float v1) {
  return p0 + (isolevel - v0) / (v1 - v0) * (p1 - p0);
}
__device__ glm::vec3 normal_interp(float isolevel, glm::vec3 n0, glm::vec3 n1, float v0, float v1) {
  return glm::normalize(n0 + (isolevel - v0) / (v1 - v0) * (n1 - n0));
}

__device__ int generate_face(int i, float isolevel) {
  int cube_idx = 0;
  int idx = i;
  int x = idx / (grid_size.y * grid_size.z);
  idx %= grid_size.y * grid_size.z;
  int y = idx / grid_size.z;
  int z = idx % grid_size.z;

  glm::vec3 vertlist[12];
  glm::vec3 normlist[12];
  float val[8];
  val[0] = get_corner_value(glm::ivec3 (x,     y,     z     ) );
  val[1] = get_corner_value(glm::ivec3 (x + 1, y,     z     ) );
  val[2] = get_corner_value(glm::ivec3 (x + 1, y,     z + 1 ) );
  val[3] = get_corner_value(glm::ivec3 (x,     y,     z + 1 ) );
  val[4] = get_corner_value(glm::ivec3 (x,     y + 1, z     ) );
  val[5] = get_corner_value(glm::ivec3 (x + 1, y + 1, z     ) );
  val[6] = get_corner_value(glm::ivec3 (x + 1, y + 1, z + 1 ) );
  val[7] = get_corner_value(glm::ivec3 (x,     y + 1, z + 1 ) );

  glm::vec3 p[8];
  glm::vec3 pn[8];
  p[0] = glm::vec3(x,   y,   z   ) * cell_size + mc_corner;
  p[1] = glm::vec3(x+1, y,   z   ) * cell_size + mc_corner;
  p[2] = glm::vec3(x+1, y,   z+1 ) * cell_size + mc_corner;
  p[3] = glm::vec3(x,   y,   z+1 ) * cell_size + mc_corner;
  p[4] = glm::vec3(x,   y+1, z   ) * cell_size + mc_corner;
  p[5] = glm::vec3(x+1, y+1, z   ) * cell_size + mc_corner;
  p[6] = glm::vec3(x+1, y+1, z+1 ) * cell_size + mc_corner;
  p[7] = glm::vec3(x,   y+1, z+1 ) * cell_size + mc_corner;

  pn[0] = get_vert_normal(glm::vec3(x,   y,   z   ));
  pn[1] = get_vert_normal(glm::vec3(x+1, y,   z   ));
  pn[2] = get_vert_normal(glm::vec3(x+1, y,   z+1 ));
  pn[3] = get_vert_normal(glm::vec3(x,   y,   z+1 ));
  pn[4] = get_vert_normal(glm::vec3(x,   y+1, z   ));
  pn[5] = get_vert_normal(glm::vec3(x+1, y+1, z   ));
  pn[6] = get_vert_normal(glm::vec3(x+1, y+1, z+1 ));
  pn[7] = get_vert_normal(glm::vec3(x,   y+1, z+1 ));

  if (val[0] > isolevel) cube_idx |= 1;
  if (val[1] > isolevel) cube_idx |= 2;
  if (val[2] > isolevel) cube_idx |= 4;
  if (val[3] > isolevel) cube_idx |= 8;
  if (val[4] > isolevel) cube_idx |= 16;
  if (val[5] > isolevel) cube_idx |= 32;
  if (val[6] > isolevel) cube_idx |= 64;
  if (val[7] > isolevel) cube_idx |= 128;

  if (edgeTable[cube_idx] == 0)
    return 0;
  if (edgeTable[cube_idx] & 1) {
    vertlist[0] = vertex_interp(isolevel,  p[0], p[1], val[0], val[1]);
    normlist[0] = normal_interp(isolevel,  pn[0], pn[1], val[0], val[1]);
  }
  if (edgeTable[cube_idx] & 2) {
    vertlist[ 1] = vertex_interp(isolevel, p[1], p[2], val[1], val[2]);
    normlist[ 1] = normal_interp(isolevel, pn[1], pn[2], val[1], val[2]);
  }
  if (edgeTable[cube_idx] & 4) {
    vertlist[ 2] = vertex_interp(isolevel, p[2], p[3], val[2], val[3]);
    normlist[ 2] = normal_interp(isolevel, pn[2], pn[3], val[2], val[3]);
  }
  if (edgeTable[cube_idx] & 8) {
    vertlist [3] = vertex_interp(isolevel, p[3], p[0], val[3], val[0]);
    normlist [3] = normal_interp(isolevel, pn[3], pn[0], val[3], val[0]);
  }
  if (edgeTable[cube_idx] & 16) {
    vertlist[ 4] = vertex_interp(isolevel, p[4], p[5], val[4], val[5]);
    normlist[ 4] = normal_interp(isolevel, pn[4], pn[5], val[4], val[5]);
  }
  if (edgeTable[cube_idx] & 32) {
    vertlist[ 5] = vertex_interp(isolevel, p[5], p[6], val[5], val[6]);
    normlist[ 5] = normal_interp(isolevel, pn[5], pn[6], val[5], val[6]);
  }
  if (edgeTable[cube_idx] & 64) {
    vertlist[ 6] = vertex_interp(isolevel, p[6], p[7], val[6], val[7]);
    normlist[ 6] = normal_interp(isolevel, pn[6], pn[7], val[6], val[7]);
  }
  if (edgeTable[cube_idx] & 128) {
    vertlist[ 7] = vertex_interp(isolevel, p[7], p[4], val[7], val[4]);
    normlist[ 7] = normal_interp(isolevel, pn[7], pn[4], val[7], val[4]);
  }
  if (edgeTable[cube_idx] & 256) {
    vertlist[ 8] = vertex_interp(isolevel, p[0], p[4], val[0], val[4]);
    normlist[ 8] = normal_interp(isolevel, pn[0], pn[4], val[0], val[4]);
  }
  if (edgeTable[cube_idx] & 512) {
    vertlist[ 9] = vertex_interp(isolevel, p[1], p[5], val[1], val[5]);
    normlist[ 9] = normal_interp(isolevel, pn[1], pn[5], val[1], val[5]);
  }
  if (edgeTable[cube_idx] & 1024) {
    vertlist[10] = vertex_interp(isolevel, p[2], p[6], val[2], val[6]);
    normlist[10] = normal_interp(isolevel, pn[2], pn[6], val[2], val[6]);
  }
  if (edgeTable[cube_idx] & 2048) {
    vertlist[11] = vertex_interp(isolevel, p[3], p[7], val[3], val[7]);
    normlist[11] = normal_interp(isolevel, pn[3], pn[7], val[3], val[7]);
  }

  int vi;
  for (vi = 0; triTable[cube_idx][vi] != -1; vi += 1) {
    faces[15 * i + vi] = vertlist[triTable[cube_idx][vi]];
    face_normals[15 * i + vi] = normlist[triTable[cube_idx][vi]];
  }
  return vi / 3;
}

__global__ void _update_faces(int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    num_faces[i] = generate_face(i, 0.2f);
  }
}

__device__ int total_num_faces;
__global__ void _transfer_faces_to_vbo(int n, float* vbo, int max_num_faces) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    int j = 0;
    if (grid_face_idx[i] + num_faces[i] >= max_num_faces) {
      if (grid_face_idx[i] < max_num_faces) {
        // printf("Warning too many faces!\n");
        total_num_faces = grid_face_idx[i];
      }
      return;
    }
    for (int t = 0; t < num_faces[i]; ++t) {
      glm::vec3 v1 = faces[15 * i + 3 * t    ];
      glm::vec3 v2 = faces[15 * i + 3 * t + 1];
      glm::vec3 v3 = faces[15 * i + 3 * t + 2];
      glm::vec3 vn1 = face_normals[15 * i + 3 * t    ];
      glm::vec3 vn2 = face_normals[15 * i + 3 * t + 1];
      glm::vec3 vn3 = face_normals[15 * i + 3 * t + 2];

      // vertex
      vbo[18 * grid_face_idx[i] + j++] = v1.x;
      vbo[18 * grid_face_idx[i] + j++] = v1.y;
      vbo[18 * grid_face_idx[i] + j++] = v1.z;

      // normal
      vbo[18 * grid_face_idx[i] + j++] = vn1.x;
      vbo[18 * grid_face_idx[i] + j++] = vn1.y;
      vbo[18 * grid_face_idx[i] + j++] = vn1.z;

      // vertex
      vbo[18 * grid_face_idx[i] + j++] = v2.x;
      vbo[18 * grid_face_idx[i] + j++] = v2.y;
      vbo[18 * grid_face_idx[i] + j++] = v2.z;

      // normal
      vbo[18 * grid_face_idx[i] + j++] = vn2.x;
      vbo[18 * grid_face_idx[i] + j++] = vn2.y;
      vbo[18 * grid_face_idx[i] + j++] = vn2.z;

      // vertex
      vbo[18 * grid_face_idx[i] + j++] = v3.x;
      vbo[18 * grid_face_idx[i] + j++] = v3.y;
      vbo[18 * grid_face_idx[i] + j++] = v3.z;

      // normal
      vbo[18 * grid_face_idx[i] + j++] = vn3.x;
      vbo[18 * grid_face_idx[i] + j++] = vn3.y;
      vbo[18 * grid_face_idx[i] + j++] = vn3.z;
    }
  }
  if (i == n - 1) {
    total_num_faces = grid_face_idx[i] + num_faces[i];
  }
}

void update_faces(float* vbo, int* h_total_num_faces, int max_num_faces) {
  int total_num_cells = h_grid_size.x * h_grid_size.y * h_grid_size.z;
  _update_faces<<<(total_num_cells + 512 - 1) / 512, 512>>>(total_num_cells);

  // stream compaction
  thrust::device_ptr<int> num_faces_ptr = thrust::device_pointer_cast(d_num_faces);
  thrust::device_ptr<int> grid_face_idx_ptr = thrust::device_pointer_cast(d_grid_face_idx);
  thrust::exclusive_scan(num_faces_ptr, num_faces_ptr + total_num_cells, grid_face_idx_ptr);

  _transfer_faces_to_vbo<<<(total_num_cells + N_THREADS - 1) / N_THREADS, N_THREADS>>>(total_num_cells, vbo, max_num_faces);
  CUDA_CHECK_RETURN(hipMemcpyFromSymbol(h_total_num_faces, HIP_SYMBOL(mc::total_num_faces), sizeof(int)));
}

void update_grid_corners() {
  hipMemset(d_grid_occupied, 0, h_grid_size.x * h_grid_size.y * h_grid_size.z * sizeof(int));
  hipMemset(d_corner_value, 0, h_corner_size.x * h_corner_size.y * h_corner_size.z * sizeof(float));
  _update_grid_corners<<<(num_particles + N_THREADS - 1) / N_THREADS, N_THREADS>>>(num_particles, h_params.cell_size);
}


void init(float cell_size) {
  h_cell_size = cell_size;
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::cell_size), &h_cell_size, sizeof(float)));

  h_mc_corner = h_domain.corner - 4 * cell_size;
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::mc_corner), &h_mc_corner, sizeof(glm::vec3)));
  h_mc_size = h_domain.size + 8 * cell_size;
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::mc_size), &h_mc_size, sizeof(glm::vec3)))

  h_grid_size = glm::ivec3(h_mc_size / cell_size) + glm::ivec3(1, 1, 1);

  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::grid_size), &h_grid_size, sizeof(glm::ivec3)));

  h_corner_size = h_grid_size + glm::ivec3(1, 1, 1);
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::corner_size), &h_corner_size, sizeof(glm::ivec3)));

  // mc grid for corners and cells
  int total_grid_size = h_grid_size.x * h_grid_size.y * h_grid_size.z;
  CUDA_CHECK_RETURN(hipMalloc(&d_grid_occupied, total_grid_size * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::grid_occupied), &d_grid_occupied, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_corner_value, h_corner_size.x * h_corner_size.y * h_corner_size.z * sizeof(float)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::corner_value), &d_corner_value, sizeof(void *), 0,
                                       hipMemcpyHostToDevice));

  // initialize face storage
  CUDA_CHECK_RETURN(hipMalloc(&d_faces, total_grid_size * 15 * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::faces), &d_faces, sizeof(void *), 0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_face_normals, total_grid_size * 15 * sizeof(glm::vec3)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::face_normals), &d_face_normals, sizeof(void *), 0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_num_faces, total_grid_size * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::num_faces), &d_num_faces, sizeof(void *), 0, hipMemcpyHostToDevice));

  CUDA_CHECK_RETURN(hipMalloc(&d_grid_face_idx, total_grid_size * sizeof(int)));
  CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(mc::grid_face_idx), &d_grid_face_idx, sizeof(void *), 0, hipMemcpyHostToDevice));
}

/*
 * idx: idx in grid_array
 * Return: center of grid
 */
glm::vec3 get_grid_center(int idx) {
  int x = idx / (h_grid_size.y * h_grid_size.z);
  idx %= h_grid_size.y * h_grid_size.z;
  int y = idx / h_grid_size.z;
  int z = idx % h_grid_size.z;
  return h_mc_corner + mc::h_cell_size * glm::vec3(x,y,z) + mc::h_cell_size / 2;
}

void print_summary() {
  log("Marching Cube");
  log("Grid size", h_grid_size);
  log("Corner size", h_corner_size);
}

int get_num_cells() {
  return h_grid_size.x * h_grid_size.y * h_grid_size.z;
}

} // namespace mc
} // namespace sph
